#pragma G++ optimize(0)
#include <hip/hip_runtime.h>
#include <random>
#include <algorithm>
#include <vector>
#include <iostream>
#include <ctime>
using namespace std;

#define N (1 << 10)
#define GRID_SIZE (1 << 7)
#define BLOCK_SIZE (1 << 6)

__global__ void gemm_baseline(float *A, float *B, float *C);
void gemm_verify(float *A, float *B, float *C);
float random_float();

int main(void) {
    std::srand(unsigned(std::time(nullptr)));
    std::vector<float> A(N*N);
    std::vector<float> B(N*N);
    std::vector<float> C(N*N, 0.0);
    std::generate(A.begin(), A.end(), random_float);
    std::generate(B.begin(), B.end(), random_float);
    dim3 grid_size(GRID_SIZE, GRID_SIZE);
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);


    float *A_cuda, *B_cuda, *C_cuda;
    if (hipMalloc((void **)&A_cuda, N*N*sizeof(float)) != hipSuccess) {
        cout << "hipMalloc failed" << endl;
        return -1;
    }
    if (hipMalloc((void **)&B_cuda, N*N*sizeof(float)) != hipSuccess) {
        cout << "hipMalloc failed" << endl;
        return -1;
    }
    if (hipMalloc((void **)&C_cuda, N*N*sizeof(float)) != hipSuccess) {
        cout << "hipMalloc failed" << endl;
        return -1;
    }
    if (hipMemcpy(A_cuda, A.data(), N*N*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        cout << "hipMemcpy failed" << endl;
        return -1;
    }
    if (hipMemcpy(B_cuda, B.data(), N*N*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        cout << "hipMemcpy failed" << endl;
        return -1;
    }

    hipEvent_t cuda_start, cuda_stop;
    float gpu_time = 0.0;
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_stop);
    hipEventRecord(cuda_start, 0);

    gemm_baseline <<<grid_size, block_size>>> (A_cuda, B_cuda, C_cuda);
    hipDeviceSynchronize();

    hipEventRecord(cuda_stop, 0);
    hipEventSynchronize(cuda_stop);
    hipEventElapsedTime(&gpu_time, cuda_start, cuda_stop);
    hipEventDestroy(cuda_start);
    hipEventDestroy(cuda_stop);

    if (hipMemcpy(C.data(), C_cuda, N*N*sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        cout << "result hipMemcpy failed" << endl;
        return -1;
    }
    hipFree(A_cuda);
    hipFree(B_cuda);
    hipFree(C_cuda);

    gemm_verify(A.data(), B.data(), C.data());
    cout << "time: " << gpu_time << endl;
    return 0;
}

float random_float(){
    static std::random_device rd;
    static std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0, 1);
    return dis(gen);
}

void gemm_verify(float *A, float *B, float *C) {
    float temp;
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            temp = 0.0;
            for(int k=0; k<N; k++){
                temp += A[i*N+k] * B[k*N+j];
            }
            if(temp-C[i*N+j] > 0.0001 || temp-C[i*N+j] < -0.0001) {
                cout << "Verify failed at " << "(" << i << ", " << j << ")" << endl;
                cout << temp << endl;
                cout << C[i*N+j] << endl;
                return;
            }
        }
    }
    cout << "Verify success!" << endl;
}


__global__ void gemm_baseline(float* A, float * B, float* C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float temp = 0;
    if((i < N) && (j < N)){
        for(int k=0; k<N; k++){
            temp += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = temp;
    }
}